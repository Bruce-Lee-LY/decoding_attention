#include "hip/hip_runtime.h"
// Copyright 2023. All Rights Reserved.
// Author: Bruce-Lee-LY
// Date: 21:14:13 on Tue, Oct 31, 2023
//
// Description: decoding fwd hdim256 and hdimv256

#include "decoding_attn/decoding_fwd_launch_template.h"

template <>
void run_dmha_fwd_<half, 256, 256>(const DecodingParams &params) {
    dmha_fwd<half, 256, 256, 256, 16>(params);
}

template <>
void run_dmha_fwd_<__hip_bfloat16, 256, 256>(const DecodingParams &params) {
    dmha_fwd<__hip_bfloat16, 256, 256, 256, 16>(params);
}
