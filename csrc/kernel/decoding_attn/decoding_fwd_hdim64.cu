#include "hip/hip_runtime.h"
// Copyright 2023. All Rights Reserved.
// Author: Bruce-Lee-LY
// Date: 21:14:13 on Tue, Oct 31, 2023
//
// Description: decoding fwd hdim64

#include "decoding_attn/decoding_fwd_launch_template.h"

template <>
void run_dmha_fwd_<half, 64>(const DecodingParams &params) {
    dmha_fwd<half, 64, 256, 4>(params);
}

template <>
void run_dmha_fwd_<__hip_bfloat16, 64>(const DecodingParams &params) {
    dmha_fwd<__hip_bfloat16, 64, 256, 4>(params);
}
